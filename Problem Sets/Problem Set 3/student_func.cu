#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <stdio.h>

#define min(a,b) (((a)<(b))?(a):(b))
#define max(a,b) (((a)>(b))?(a):(b))

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in,
                                    int n, int op)
{
  // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid  = threadIdx.x;

  if (myId >= n)
    return;

  // load shared mem from global mem
  sdata[tid] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      if (op == 0)
        sdata[tid] = min(sdata[tid], sdata[tid + s]);
      else if (op == 1)
        sdata[tid] = max(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
      d_out[blockIdx.x] = sdata[0];
  }
}

__global__ void histogram(const float * const d_logLuminance, 
                          unsigned int *d_hist,
                          float lumMin, 
                          const float range,
                          const int numBins)
{
  int myId     = threadIdx.x + blockIdx.x * blockDim.x;
  float lum    = d_logLuminance[myId];
  int myBin    = (lum - lumMin) / range * numBins;
  atomicAdd(&(d_hist[myBin]), 1);
}

__global__ void exclusiveScan(unsigned int * d_hist, 
                              unsigned int * const d_cdf, 
                              const int numBins)
{
  extern __shared__ unsigned int tmp[];
  int tid = threadIdx.x;
  
  tmp[tid] = (tid > 0) ? d_hist[tid - 1] : 0;
  __syncthreads();
  
  for(int s = 1; s < numBins; s *= 2)
  { 
    unsigned int t = tmp[tid];
    __syncthreads();
    
    if(tid + s < numBins)
    { 
      tmp[tid + s] += t;
    } 
    __syncthreads();
  }
  d_cdf[tid] = tmp[tid];
}


void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /*1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum */
  int n = numRows * numCols;

  // declare GPU memory pointers
  float * d_intermediate;
  float * d_temp;

  // allocate GPU memory
  checkCudaErrors(hipMalloc((void **) &d_intermediate, n*sizeof(float)));
  checkCudaErrors(hipMalloc((void **) &d_temp, sizeof(float)));

  const int maxThreadsPerBlock = 1024;
  int threads = maxThreadsPerBlock;
  int blocks  = n / maxThreadsPerBlock;
  int shared  = threads * sizeof(float);

  shmem_reduce_kernel<<<blocks, threads, shared>>>
      (d_intermediate, d_logLuminance, n, 0);

  shmem_reduce_kernel<<<1, blocks, shared>>>
      (d_temp, d_intermediate, n, 0);

  checkCudaErrors(hipMemcpy(&min_logLum, d_temp, sizeof(float),
                  hipMemcpyDeviceToHost));

  shmem_reduce_kernel<<<blocks, threads, shared>>>
      (d_intermediate, d_logLuminance, n, 1);

  shmem_reduce_kernel<<<1, blocks, shared>>>
      (d_temp, d_intermediate, n, 1);

  checkCudaErrors(hipMemcpy(&max_logLum, d_temp, sizeof(float),
                  hipMemcpyDeviceToHost));

  /*2) subtract them to find the range */
  float range = max_logLum - min_logLum;

  /*3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins */
  unsigned int * d_hist, * h_hist;

  checkCudaErrors(hipMalloc((void**) &d_hist, sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_hist, 0, sizeof(int)*numBins));

  histogram<<<blocks, threads>>>
      (d_logLuminance, d_hist, min_logLum, range, numBins);

  /*4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
  exclusiveScan<<<1, threads, sizeof(unsigned int) * threads>>>
      (d_hist, d_cdf, numBins);
}


